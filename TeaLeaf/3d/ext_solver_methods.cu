#include "hip/hip_runtime.h"
#include <cstdio>
#include "ext_cuda_chunk.hpp"
#include "kernels/solver_methods.cuknl"

/*
 *		SHARED SOLVER METHODS
 */

// Entry point to copy U.
extern "C"
void ext_solver_copy_u_(
		const int* chunk)
{
	Chunks[*chunk-1]->CopyU();
}

// Entry point for calculating residual.
extern "C"
void ext_calculate_residual_(
		const int* chunk)
{
	Chunks[*chunk-1]->CalculateResidual();
}

// Entry point for calculating 2norm.
extern "C"
void ext_calculate_2norm_(
		const int* chunk,
		const int* normArray,
	   	double* normOut)
{
	Chunks[*chunk-1]->Calculate2Norm(*normArray, normOut);
}

// Entry point for finalising solution.
extern "C"
void ext_solver_finalise_(
		const int* chunk)
{
	Chunks[*chunk-1]->Finalise();
}

// Determines the rx, ry and rz values.
void TeaLeafCudaChunk::CalcRxRyRz(
		const double dt,
		double* rxOut,
		double* ryOut,
		double* rzOut)
{
	double dx, dy, dz;

	hipMemcpy(&dx, dCellDx, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&dy, dCellDy, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&dz, dCellDz, sizeof(double), hipMemcpyDeviceToHost);
	TeaLeafCudaChunk::CheckErrors(__LINE__,__FILE__);

	*rxOut = dt/(dx*dx);
	*ryOut = dt/(dy*dy);
	*rzOut = dt/(dz*dz);
}

// Copies the current value of u
void TeaLeafCudaChunk::CopyU()
{
	PRE_KERNEL(2*HALO_PAD);

	CuKnlCopyU<<<numBlocks, BLOCK_SIZE>>>(
			innerX, innerY, innerZ, xCells, xCells*yCells, dU, dU0);

	POST_KERNEL("Copy U");
}

// Calculates the current residual value.
void TeaLeafCudaChunk::CalculateResidual()
{
	PRE_KERNEL(2*HALO_PAD);

	CuKnlCalculateResidual<<<numBlocks, BLOCK_SIZE>>>(
			innerX, innerY, innerZ, xCells, xCells*yCells, 
			dU, dU0, dKx, dKy, dKz, dR);

	POST_KERNEL("Calculate Residual");
}

// Calculates the 2norm of a particular space.
void TeaLeafCudaChunk::Calculate2Norm(
		const bool normArray,
		double* normOut)
{
	PRE_KERNEL(2*HALO_PAD);

	CuKnlCalculate2Norm<<<numBlocks, BLOCK_SIZE>>>(
			innerX, innerY, innerZ, xCells, xCells*yCells, 
			normArray ? dR : dU0, dReduceBuffer1);

	POST_KERNEL("Calculate 2Norm");
	SumReduce(dReduceBuffer1, normOut, numBlocks, "2norm reduction");
}

// Reduces residual values of a buffer
void TeaLeafCudaChunk::SumReduce(
		double* buffer,
		double* result,
		int len,
		std::string kName)
{
	while(len > 1)
	{
		int numBlocks = ceil(len/(float)BLOCK_SIZE);
		START_PROFILING();
		CuKnlSumReduce<<<numBlocks,BLOCK_SIZE>>>(len, buffer);
		POST_KERNEL(kName);
		len = numBlocks;
	}

	hipMemcpy(result, buffer, sizeof(double), hipMemcpyDeviceToHost);
	CheckErrors(__LINE__,__FILE__);
}

// Finalises the solution.
void TeaLeafCudaChunk::Finalise()
{
	PRE_KERNEL(2*HALO_PAD);

	CuKnlFinalise<<<numBlocks, BLOCK_SIZE>>>(
			innerX, innerY, innerZ, xCells, xCells*yCells, dDensity, dU, dEnergy1);

	POST_KERNEL("Finalise Solver");
}

// Loads alphas and betas onto the device
void TeaLeafCudaChunk::LoadAlphaBeta(
		const double* alphas,
		const double* betas,
		const int numCoefs)
{
	size_t length = numCoefs*sizeof(double);
	hipMalloc((void**) &dAlphas, length);
	hipMalloc((void**) &dBetas, length);
	hipMemcpy(dAlphas, alphas, length, hipMemcpyHostToDevice);
	hipMemcpy(dBetas, betas, length, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	CheckErrors(__LINE__,__FILE__);
}
