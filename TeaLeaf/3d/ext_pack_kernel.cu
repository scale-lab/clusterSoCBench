#include "hip/hip_runtime.h"
#include <cstdio>
#include <numeric>
#include <iostream>
#include "ext_cuda_chunk.hpp"
#include "kernels/pack_kernel.cuknl"

using std::ceil;
using std::accumulate;

#define CELL_DATA 1
#define VERTEX_DATA 2
#define X_FACE_DATA 3
#define Y_FACE_DATA 4
#define Z_FACE_DATA 5
#define WARP_SIZE 32.0

#define PREPARE_INC(fieldType) \
	int xInc = 0; \
	int yInc = 0; \
	int zInc = 0; \
	switch(fieldType) \
	{ \
		case CELL_DATA:\
			break; \
		case VERTEX_DATA: \
			xInc = yInc = zInc = 1; \
			break; \
		case X_FACE_DATA: \
			xInc = 1; \
			break; \
		case Y_FACE_DATA: \
			yInc = 1; \
			break; \
		case Z_FACE_DATA: \
			zInc = 1; \
			break; \
	} \

// Entry point for packing messages
extern "C"
void ext_pack_message_(
		const int* chunk,
		const int* fields,
		const int* offsets,
		const int* depth,
		const int* face,
		const int* fieldType,
		double* buffer)
{
	Chunks[*chunk-1]->PackUnpackKernel(fields, offsets, *depth, *face, *fieldType, buffer, true);
}

// Entry point for unpacking messages
extern "C"
void ext_unpack_message_(
		const int* chunk,
		const int* fields,
		const int* offsets,
		const int* depth,
		const int* face,
		const int* fieldType,
		double* buffer)
{
	Chunks[*chunk-1]->PackUnpackKernel(fields, offsets, *depth, *face, *fieldType, buffer, false);
}

// Performs buffer packing and unpacking
void TeaLeafCudaChunk::PackUnpackKernel(
		const int* fields,
		const int* offsets,
		const int depth,
		const int face,
		const int fieldType,
		double* buffer,
		const bool pack)
{
	const int exchanges = accumulate(fields, fields+NUM_FIELDS, 0);

	if(exchanges < 1) return;

	PREPARE_INC(fieldType);

	std::string kernelName;
	double* deviceBuffer = NULL;
	CuKnlPackType packKernel = NULL;

	int bufferLength = 0;
	int innerX = xCells-2*HALO_PAD;
	int innerY = yCells-2*HALO_PAD;
	int innerZ = zCells-2*HALO_PAD;
	
	switch(face)
	{
		case CHUNK_LEFT:
			kernelName = (pack) ? "Pack Left" : "Unpack Left";
			packKernel = (pack) ? CuKnlPackLeft : CuKnlUnpackLeft;
			deviceBuffer = dLeftBuffer;
			bufferLength = innerY*innerZ*depth;
			break;
		case CHUNK_RIGHT:
			kernelName = (pack) ? "Pack Right" : "Unpack Right";
			packKernel = (pack) ? CuKnlPackRight : CuKnlUnpackRight;
			deviceBuffer = dRightBuffer;
			bufferLength = innerY*innerZ*depth;
			break;
		case CHUNK_TOP:
			kernelName = (pack) ? "Pack Top" : "Unpack Top";
			packKernel = (pack) ? CuKnlPackTop : CuKnlUnpackTop;
			deviceBuffer = dTopBuffer;
			bufferLength = innerX*innerZ*depth;
			break;
		case CHUNK_BOTTOM:
			kernelName = (pack) ? "Pack Bottom" : "Unpack Bottom";
			packKernel = (pack) ? CuKnlPackBottom : CuKnlUnpackBottom;
			deviceBuffer = dBottomBuffer;
			bufferLength = innerX*innerZ*depth;
			break;
		case CHUNK_FRONT:
			kernelName = (pack) ? "Pack Front" : "Unpack Front";
			packKernel = (pack) ? CuKnlPackFront : CuKnlUnpackFront;
			deviceBuffer = dFrontBuffer;
			bufferLength = innerX*innerY*depth;
			break;
		case CHUNK_BACK:
			kernelName = (pack) ? "Pack Back" : "Unpack Back";
			packKernel = (pack) ? CuKnlPackBack : CuKnlUnpackBack;
			deviceBuffer = dBackBuffer;
			bufferLength = innerX*innerY*depth;
			break;
		default:
			TeaLeafCudaChunk::Abort(__LINE__, __FILE__, 
					"Incorrect face provided: %d.\n", face);
	}

	if(!pack)
	{
		hipMemcpy(deviceBuffer, buffer, exchanges*bufferLength*sizeof(double), 
				hipMemcpyHostToDevice);
		TeaLeafCudaChunk::CheckErrors(__LINE__,__FILE__);
	}

	int offset = 0;
	int numBlocks = ceil(bufferLength/(float)BLOCK_SIZE);

	for(int ii = 0; ii != NUM_FIELDS; ++ii)
	{
		if(fields[ii])
		{
			double* deviceField = NULL;
			switch(ii+1)
			{
				case FIELD_DENSITY:
					deviceField = dDensity;
					break;
				case FIELD_ENERGY0:
					deviceField = dEnergy0;
					break;
				case FIELD_ENERGY1:
					deviceField = dEnergy1;
					break;
				case FIELD_U:
					deviceField = dU;
					break;
				case FIELD_P:
					deviceField = dP;
					break;
				case FIELD_SD:
					deviceField = dSd;
					break;
				default:
					TeaLeafCudaChunk::Abort(__LINE__,__FILE__,
							"Incorrect field provided: %d.\n", ii+1);
			}

			START_PROFILING();

			int bufferOffset = bufferLength*offset++;
			packKernel<<<numBlocks, BLOCK_SIZE>>>(
					xCells, yCells, zCells, innerX, innerY, innerZ, 
					deviceField, deviceBuffer+bufferOffset, depth);

			POST_KERNEL(kernelName.c_str());
		}
	}

	if(pack)
	{
		hipMemcpy(buffer, deviceBuffer, exchanges*bufferLength*sizeof(double),
				hipMemcpyDeviceToHost);
		TeaLeafCudaChunk::CheckErrors(__LINE__,__FILE__);
	}
}
