#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include "ext_cuda_chunk.hpp"
#include "kernels/update_halo.cuknl"

/*
 * 		UPDATE HALO KERNEL
 */	

using std::ceil;

// Entry point for update the halo
extern "C"
void ext_update_halo_kernel_(
		const int* chunk,
		const int* chunkNeighbours,
		const int* fields,
		const int* depth)
{
	Chunks[*chunk-1]->UpdateHalo(chunkNeighbours, fields, *depth);
}

// Updates all necessary fields for a particular halo
void TeaLeafCudaChunk::UpdateHalo(
		const int* chunkNeighbours,
		const int* fields,
		const int depth)
{
#define LAUNCH_UPDATE(index, buffer, depth)\
	if(fields[index-1])\
	{\
		UpdateFace(chunkNeighbours, depth, buffer);\
	}

	LAUNCH_UPDATE(FIELD_P, dP, depth);
	LAUNCH_UPDATE(FIELD_DENSITY, dDensity, depth);
	LAUNCH_UPDATE(FIELD_ENERGY0, dEnergy0, depth);
	LAUNCH_UPDATE(FIELD_ENERGY1, dEnergy1, depth);
	LAUNCH_UPDATE(FIELD_U, dU, depth);
	LAUNCH_UPDATE(FIELD_SD, dSd, depth);
}

// Updates a field for each required face of a halo
void TeaLeafCudaChunk::UpdateFace(
		const int* chunkNeighbours,
		const int depth,
		double* buffer)
{
	int innerX = xCells-HALO_PAD*2; 
	int innerY = yCells-HALO_PAD*2;
	int innerZ = zCells-HALO_PAD*2;

#define UPDATE_FACE(face, kernelName, updateKernel) \
	if(chunkNeighbours[face-1] == EXTERNAL_FACE)\
	{\
		START_PROFILING();\
		updateKernel<<<numBlocks, BLOCK_SIZE>>>(\
				xCells, yCells, zCells, innerX, \
				innerY, innerZ, depth, buffer);\
		POST_KERNEL(kernelName);\
	}

	int numBlocks = ceil((innerX*innerZ*depth)/(float)BLOCK_SIZE);
	UPDATE_FACE(CHUNK_TOP, "Halo Top", CuKnlUpdateTop);
	UPDATE_FACE(CHUNK_BOTTOM, "Halo Bottom", CuKnlUpdateBottom);

	numBlocks = ceil((innerX*innerY*depth)/(float)BLOCK_SIZE);
	UPDATE_FACE(CHUNK_BACK, "Halo Back", CuKnlUpdateBack);
	UPDATE_FACE(CHUNK_FRONT, "Halo Front", CuKnlUpdateFront);

	numBlocks = ceil((innerY*innerZ*depth)/(float)BLOCK_SIZE);
	UPDATE_FACE(CHUNK_RIGHT, "Halo Right", CuKnlUpdateRight);
	UPDATE_FACE(CHUNK_LEFT, "Halo Left", CuKnlUpdateLeft);
}
