#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>
#include "ext_cuda_chunk.hpp"
#include "kernels/jacobi_solve.cuknl"

/*
 *		JACOBI SOLVER KERNEL
 */

using std::ceil;

// Entry point for Jacobi initialisation.
extern "C"
void ext_jacobi_kernel_init_(
		const int* chunk,
		const int* coefficient,
		const double* dt,
		double* rx,
		double* ry,
		double* rz)
{
	Chunks[*chunk-1]->JacobiInit(*dt, rx, ry, rz, *coefficient);
}

// Entry point for Jacobi solver main method.
extern "C"
void ext_jacobi_kernel_solve_(
		const int* chunk,
		double* error)
{
	Chunks[*chunk-1]->JacobiSolve(error);
}

// Jacobi solver initialisation method.
void TeaLeafCudaChunk::JacobiInit(
		const double dt,
		double* rx,
		double* ry,
		double* rz,
		const int coefficient)
{
	if(coefficient != CONDUCTIVITY && coefficient != RECIP_CONDUCTIVITY)
	{
		Abort(__LINE__, __FILE__, "Coefficient %d is not valid.\n", coefficient);
	}

	CalcRxRyRz(dt, rx, ry, rz);

	PRE_KERNEL(HALO_PAD);

	CuKnlJacobiInit<<<numBlocks, BLOCK_SIZE>>>(
			innerX, innerY, innerZ, xCells, xCells*yCells, 
			dDensity, dEnergy1, *rx, *ry, *rz, dKx, dKy, 
			dKz, dU0, dU, coefficient);

	POST_KERNEL("Jacobi Initialise");
}

void TeaLeafCudaChunk::JacobiCopyU()
{
	PRE_KERNEL(0);

	CuKnlJacobiCopyU<<<numBlocks, BLOCK_SIZE>>>(
			innerX, innerY, innerZ, dU, dR);

	POST_KERNEL("Jacobi Copy U");
}

// Main Jacobi solver method.
void TeaLeafCudaChunk::JacobiSolve(
		double* error)
{
	JacobiCopyU();

	PRE_KERNEL(2*HALO_PAD);

	CuKnlJacobiSolve<<<numBlocks, BLOCK_SIZE>>>(
			innerX, innerY, innerZ, xCells, xCells*yCells, 
			dKx, dKy, dKz, dU0, dR, dU, dReduceBuffer1);

	POST_KERNEL("Jacobi Solve");

	SumReduce(dReduceBuffer1, error, numBlocks, "Jacobi Reduction");
}

