#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>
#include "ext_cuda_chunk.hpp"
#include "kernels/ppcg_solve.cuknl"

/*
 *		PPCG SOLVER KERNEL
 */

using std::ceil;

// Entry point for PPCG initialisation
extern "C"
void ext_ppcg_init_(
		const int* chunk,
		const int* preconditionerOn,
		const double* alphas,
		const double* betas,
		int* numSteps)
{
	Chunks[*chunk-1]->PPCGInit(
			*preconditionerOn, alphas, betas, *numSteps);
}

// Entry point for initialising sd
extern "C"
void ext_ppcg_init_sd_(
		const int* chunk,
		const double* theta)
{
	Chunks[*chunk-1]->PPCGInitSd(*theta);
}

// Entry point for the PPCG inner step
extern "C"
void ext_ppcg_inner_(
		const int* chunk,
		const int* currentStep)
{
	Chunks[*chunk-1]->PPCGInner(*currentStep);
}

// Initialises the PPCG solver
void TeaLeafCudaChunk::PPCGInit(
		const bool preconditionerOn,
		const double* alphas,
		const double* betas,
		const int numSteps)
{
	preconditioner = preconditionerOn;
	LoadAlphaBeta(alphas, betas, numSteps);
}

// Initialises sd
void TeaLeafCudaChunk::PPCGInitSd(
		const double theta)
{
	PRE_KERNEL(2*HALO_PAD);

	CuKnlPPCGInitSd<<<numBlocks, BLOCK_SIZE>>>(
		innerX, innerY, innerZ, xCells, xCells*yCells,
		theta, preconditioner, dR, dMi, dSd);

	POST_KERNEL("PPCG Init Sd");
}

// The PPCG inner step
void TeaLeafCudaChunk::PPCGInner(
		const int currentStep)
{
	PRE_KERNEL(2*HALO_PAD);

	CuKnlPPCGUpdateR<<<numBlocks, BLOCK_SIZE>>>(
		innerX, innerY, innerZ, xCells, xCells*yCells, dKx,
		dKy, dKz, dSd, dU, dR);

	POST_KERNEL("PPCG Update R");

	START_PROFILING();

	CuKnlPPCGCalcSd<<<numBlocks, BLOCK_SIZE>>>(
		innerX, innerY, innerZ, xCells, xCells*yCells, currentStep-1,
		preconditioner, dR, dMi, dAlphas, dBetas, dSd);

	POST_KERNEL("PPCG Calc Sd");
}

