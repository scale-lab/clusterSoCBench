#include "hip/hip_runtime.h"
#include <cstdio>
#include <algorithm>
#include "ext_cuda_chunk.hpp"
#include "kernels/initialise_chunk.cuknl"

/*
 * 		INITIALISE CHUNK KERNEL
 */

// Extended CUDA kernel for the chunk initialisation
extern "C"
void ext_initialise_chunk_cuda_( 
		const int* chunk,
		const double* xMin,
		const double* yMin,
		const double* zMin,
		const double* dx,
		const double* dy,
		const double* dz)
{
	Chunks[*chunk-1]->InitialiseChunk(*xMin, *yMin, *zMin, *dx, *dy, *dz);
}

// Initialises the chunk's primary data fields.
void TeaLeafCudaChunk::InitialiseChunk( 
		const double xMin,
		const double yMin,
		const double zMin,
		const double dx,
		const double dy,
		const double dz)
{
	int numCells = 1+std::max(xCells, std::max(yCells, zCells));
	int numBlocks = std::ceil((float)numCells/(float)BLOCK_SIZE);

	START_PROFILING();

	CuKnlInitialiseChunkVertices<<<numBlocks, BLOCK_SIZE>>>(
			xCells, yCells, zCells, xMin, yMin, zMin, dx, dy, dz,
			dVertexX, dVertexY, dVertexZ, dVertexDx, dVertexDy, dVertexDz);

	POST_KERNEL("Initialise Chunk Vertices");

	numCells = (xCells+1)*(yCells+1)*(zCells+1);
	numBlocks = std::ceil((float)numCells/(float)BLOCK_SIZE);

	START_PROFILING();

	CuKnlInitialiseChunk<<<numBlocks, BLOCK_SIZE>>>(
			xCells, yCells, zCells, dx, dy, dz,
			dVertexX, dVertexY, dVertexZ, 
			dCellX, dCellY, dCellZ, 
			dCellDx, dCellDy, dCellDz, 
			dVolume, dXArea, dYArea, dZArea);

	POST_KERNEL("Initialise Chunk Final");
}
